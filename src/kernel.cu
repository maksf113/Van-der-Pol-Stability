#include "hip/hip_runtime.h"
﻿#include "kernel.h"
#include <stdio.h>
#include <math.h>

constexpr int TX = 32;
constexpr int TY = 32;
constexpr float len = 5.0;
constexpr float dt = 0.005;
constexpr float finalTime = 10.0;
/*****************************************************
* solving equation:
* x'' = f(x, x', t)
* e.g.: x'' + x - const * (1 - x^2) * x' for van der Pol oscialtor
* substituting y = x' results in system of equations:
* x' = y
* y' = f(x, y, t)
* 
* Descretized variables:
t_n = dt * n
* x_k = x(t_k)
* y_k = y(t_k)
* Forward Euler:
* x_k+1 = x_k + y_k * dt
* y_k+1 = y_k + f(x_k, y_k, t_k) * dt
*****************************************************/
__device__
unsigned char clip(int n) { return n > 255 ? 255 : (n < 0 ? 0 : n); }

// sclae int [-len, len]
__device__
float scale(int i, int w) { return 2.0f * len * (float(i) / w - 0.5f); }
// RHS for the equation
__device__
float f(float x, float y, float param, int sys)
{
    if (sys == 1) return x - 2 * param * y; // negative stiffness (reversed pendulum)
    if (sys == 2) return -x + param * (1.0f - x * x) * y; // van der Pol
    else return -x - 2 * param * y; // pendulum
}
// explicit Euler solver
__device__
float2 euler(float x, float y, float dt, float tFinal, float param, int sys)
{
    for (float t = 0.0f; t <= tFinal; t += dt)
    {
        x = x + y * dt;
        y = y + f(x, y, param, sys) * dt;
    }
    return make_float2(x, y);
}

__global__
void stabilityKernel(uchar4* d_out, int width, int height, float param, int sys)
{
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (col >= width || row >= height)
        return;
    // index
    const int i = row * width + col;
    // initial state
    const float x0 = scale(col, width);
    const float y0 = scale(row, height);
    // initial distance from stable solution (0,0)
    const float dist0 = sqrt(x0 * x0 + y0 * y0);
    const float2 finalPos = euler(x0, y0, dt, finalTime, param, sys);
    const float distFinal = sqrt(finalPos.x * finalPos.x + finalPos.y * finalPos.y);
    // assign color based on distance change
    const float distRatio = distFinal / dist0;

    d_out[i].x = clip(distRatio * 255); // red - growth
    d_out[i].y = (col == width / 2 || row == height / 2) ? 166 : 0; // green - axes
    d_out[i].z = clip((1.0f / distRatio) * 255); // blue - 1/growth
    d_out[i].w = 255; // alpha channel
}

void stabilityKernelLauncher(uchar4* d_out, int width, int height, float param, int sys)
{
    const dim3 blockSize = dim3(TX, TY);
    const dim3 gridSize = dim3((width + TX - 1) / TX, (height + TY - 1) / TY);
    stabilityKernel << <gridSize, blockSize >> > (d_out, width, height, param, sys);
    CUDA(hipGetLastError()); // Check for launch errors
    CUDA(hipDeviceSynchronize()); // Check for runtime errors
}


__global__
void distKernel(uchar4* d_out, int w, int h, int2 pos)
{
    const int c = blockIdx.x * blockDim.x + threadIdx.x;
    const int r = blockIdx.y * blockDim.y + threadIdx.y;
    if ((c >= w) || (r >= h))
        return;
    const int i = r * w + c;

    const int d = sqrtf((c - pos.x) * (c - pos.x) + (r - pos.y) * (r - pos.y));

    const unsigned char intensity = clip(255 - d);
    d_out[i].x = intensity;
    d_out[i].y = intensity;
    d_out[i].z = 0;
    d_out[i].w = 255;
}

void distKernelLauncher(uchar4* d_out, int w, int h, int2 pos)
{
    const dim3 blockSize(TX, TY);
    const int bx = (w + TX - 1) / TX;
    const int by = (h + TY - 1) / TY;
    const dim3 gridSize(bx, by);
    distKernel << <gridSize, blockSize >> > (d_out, w, h, pos);
    CUDA(hipGetLastError()); // Check for launch errors
    CUDA(hipDeviceSynchronize()); // Check for runtime errors
}